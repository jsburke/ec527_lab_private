#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <math.h>

#define MINVAL 0.00
#define MAXVAL 10.0

////////////////////////////  CUDA RELATED  ////////////////////////////////////

#define 

#define IMUL(a, b) __mul24(a, b)

// Assertion to check for errors
#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"CUDA_SAFE_CALL: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__global__ void SOR(float* arr, int len, float OMEGA)
{

	// start with some bounds checking to be safe
	if ((threadIdx.x >= 0) && (threadIdx.x < 15))
	{
		if ((threadIdx.y >= 0) && (threadIdx.y < 15))
		{
			int   i_start, i_end, j_start, j_end;
			float change = 0;

			// set start point for threads
			if (threadIdx.x == 0) i_start = 1;
			else				  i_start = threadIdx.x * 128;

			if (threadIdx.y == 0) j_start = 1;
			else				  j_start = threadIdx.y * 128;

			// set end point for threads
			if (threadIdx.x == 15) i_end = 2046;
			else                   i_end = threadIdx.x * 128 + 127;

			if (threadIdx.y == 15) j_end = 2046;
			else                   j_end = threadIdx.y * 128 + 127;

			
		}
	}
}

/////////////////////////////  MATRIX STUFF  ////////////////////////////////////////

float* matrix_create(int len);
void   matrix_init(float* mat, int len);

/////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char *argv[])
{
	int LENGTH;

	if(argc != 2)
	{
		printf("\n\tProgram requires input length\n");
		return 0;
	}

	LENGTH = strtol(argv[1], NULL, 10);

	if(LENGTH <= 0)
	{
		printf("\n\tLENGTH must be greater than zero\n");
		return 0;
	}

	// Create and initialize a 2D array

	float* arr = matrix_create(LENGTH);
	if(!arr) return 0;

	if(!matrix_init(arr, LENGTH))
	{
		printf("\n\tFailed to initialize matrix\n");
		return 0;
	}



	return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////  MATRIX IMPLEMENTATIONS  ////////////////////////////////////////
float float_rand(float min, float max)
{
	float f = (float)random()/RAND_MAX;
	return  min + f * (max - min);
}


float* matrix_create(int len)
{
	if(len > 0)
	{
		float* arr = (float*) calloc(len*len, sizeof(float));
		if(!arr)
		{
			printf("\n\tFailed to allocate array\n");
			return NULL;
		}
	}
	else return NULL;

	return arr;
}

int matrix_init(float* mat, int len)
{
	int len_sq, i;

	if(len > 0)
	{
		len_sq = len * len;
		for (i = 0; i < len_sq; i++)
		{
			mat[i] = float_rand(MINVAL, MAXVAL);
		}
		return 1;
	}
	return 0;
}