#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <math.h>

#define MINVAL 0.00
#define MAXVAL 10.0

////////////////////////////  CUDA RELATED  ////////////////////////////////////

#define 

#define IMUL(a, b) __mul24(a, b)

// Assertion to check for errors
#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"CUDA_SAFE_CALL: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__global__ void SOR_kernel(float* arr, int len, float OMEGA)
{

	// start with some bounds checking to be safe
	if ((threadIdx.x >= 0) && (threadIdx.x < 15))
	{
		if ((threadIdx.y >= 0) && (threadIdx.y < 15))
		{
			
			// variables needed for SOR
			int   i_start, i_end, j_start, j_end;
			float change = 0;

			// set start point for threads
			if (threadIdx.x == 0) i_start = 1;
			else				  i_start = threadIdx.x * 128;

			if (threadIdx.y == 0) j_start = 1;
			else				  j_start = threadIdx.y * 128;

			// set end point for threads
			if (threadIdx.x == 15) i_end = 2046;
			else                   i_end = threadIdx.x * 128 + 127;

			if (threadIdx.y == 15) j_end = 2046;
			else                   j_end = threadIdx.y * 128 + 127;

			//  begin the SOR this portion is responsible for

			int i,j,k;

			for (k = 0; k < 2000; k++)  //2k iterations of SOR
			{
				for (i = i_start; i <= i_end; i++)
				{
					for (j = j_start; j <= j_end; j++)
					{
						change = arr[i*len+j] - 0.25 * (arr[(i-1)*len+j] + arr[(i+1)*len+j] + arr[i*len+j+1] + arr[i*len+j-1]);

						__syncthreads();

						arr[i*len+j] -= change * OMEGA;

						__syncthreads();
					}
				}
			}
		}
	}
}

/////////////////////////////  MATRIX STUFF  ////////////////////////////////////////

float* matrix_create(int len);
void   matrix_init(float* mat, int len);

/////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char *argv[])
{
	int LENGTH = 2048;

	// Create and initialize a 2D array
	float *arr, *d_arr, *d_res, *h_res;
	int    size = LENGTH * LENGTH;

	// initialize array on host
	arr = matrix_create(LENGTH);
	if(!arr) return 0;

	if(!matrix_init(arr, LENGTH))
	{
		printf("\n\tFailed to initialize matrix\n");
		return 0;
	}

	// send array to GPU
	CUDA_SAFE_CALL(hipMalloc((void**) &d_arr, size));
	CUDA_SAFE_CALL(hipMemcpy(d_arr, arr, size, hipMemcpyHostToDevice));

	// create single block of 16x16 threads
	dim3 dimBlock(16, 16, 1);
	dim3 dimGrid(1, 1, 1);

	// invoke kernel, harcode OMEGA
	SOR_kernel<<<dimGrid, dimBlock>>>(d_arr, LENGTH, 1.97);

	// transfer back to CPU
	CUDA_SAFE_CALL(hipMemcpy(h_arr, d_arr, size, cudMemcpyDeviceToHost));

	return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////  MATRIX IMPLEMENTATIONS  ////////////////////////////////////////
float float_rand(float min, float max)
{
	float f = (float)random()/RAND_MAX;
	return  min + f * (max - min);
}


float* matrix_create(int len)
{
	if(len > 0)
	{
		float* arr = (float*) calloc(len*len, sizeof(float));
		if(!arr)
		{
			printf("\n\tFailed to allocate array\n");
			return NULL;
		}
	}
	else return NULL;

	return arr;
}

int matrix_init(float* mat, int len)
{
	int len_sq, i;

	if(len > 0)
	{
		len_sq = len * len;
		for (i = 0; i < len_sq; i++)
		{
			mat[i] = float_rand(MINVAL, MAXVAL);
		}
		return 1;
	}
	return 0;
}