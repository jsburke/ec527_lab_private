
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <math.h>

// Assertion to check for errors
#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"CUDA_SAFE_CALL: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

#define NUM_THREADS_PER_BLOCK 	256
#define NUM_BLOCKS 				16
#define PRINT_TIME 				1
#define SM_ARR_LEN				50000
#define TOL						1e-6

#define IMUL(a, b) __mul24(a, b)

void initializeArray1D(float *arr, int len, int seed);

__global__ void kernel_add (int arrLen, float* x, float* y, float* result) {
	const int tid = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;
	const int threadN = IMUL(blockDim.x, gridDim.x);
	
	int i;
	
	for(i = tid; i < arrLen; i += threadN) {
		result[i] = (1e-6 * x[i] ) + (1e-7 * y[i]) + 0.25;
	}
}

int main(int argc, char **argv){
	int arrLen = 0;
		
	// GPU Timing variables
	hipEvent_t start, stop;
	float elapsed_gpu;
	
	// Arrays on GPU global memoryc
	float *d_x;
	float *d_y;
	float *d_result;

	// Arrays on the host memory
	float *h_x;
	float *h_y;
	float *h_result;
	float *h_result_gold;
	
	int i, errCount = 0, zeroCount = 0;
	
	if (argc > 1) {
		arrLen  = atoi(argv[1]);
	}
	else {
		arrLen = SM_ARR_LEN;
	}

	printf("Length of the array = %d\n", arrLen);

    // Select GPU
    CUDA_SAFE_CALL(hipSetDevice(0));

	// Allocate GPU memory
	size_t allocSize = arrLen * sizeof(float);
	CUDA_SAFE_CALL(hipMalloc((void **)&d_x, allocSize));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_y, allocSize));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_result, allocSize));
		
	// Allocate arrays on host memory
	h_x                        = (float *) malloc(allocSize);
	h_y                        = (float *) malloc(allocSize);
	h_result                   = (float *) malloc(allocSize);
	h_result_gold              = (float *) malloc(allocSize);
	
	// Initialize the host arrays
	printf("\nInitializing the arrays ...");
	// Arrays are initialized with a known seed for reproducability
	initializeArray1D(h_x, arrLen, 2453);
	initializeArray1D(h_y, arrLen, 1467);
	printf("\t... done\n\n");
	
	
#if PRINT_TIME
	// Create the cuda events
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// Record event on the default stream
	hipEventRecord(start, 0);
#endif
	
	// Transfer the arrays to the GPU memory
	CUDA_SAFE_CALL(hipMemcpy(d_x, h_x, allocSize, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_y, h_y, allocSize, hipMemcpyHostToDevice));
	  
	// Launch the kernel
	kernel_add<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK>>>(arrLen, d_x, d_y, d_result);

	// Check for errors during launch
	CUDA_SAFE_CALL(hipPeekAtLastError());
	
	// Transfer the results back to the host
	CUDA_SAFE_CALL(hipMemcpy(h_result, d_result, allocSize, hipMemcpyDeviceToHost));
	
#if PRINT_TIME
	// Stop and destroy the timer
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_gpu, start, stop);
	printf("\nGPU time: %f (msec)\n", elapsed_gpu);
	hipEventDestroy(start);
	hipEventDestroy(stop);
#endif
	
	// Compute the results on the host
	for(i = 0; i < arrLen; i++) {
		h_result_gold[i] = (1e-6 * h_x[i]) + (1e-7 * h_y[i]) + 0.25;
	}
	
	// Compare the results
	for(i = 0; i < arrLen; i++) {
		if (abs(h_result_gold[i] - h_result[i]) > TOL) {
			errCount++;
		}
		if (h_result[i] == 0) {
			zeroCount++;
		}
	}
	
	/*
	for(i = 0; i < 50; i++) {
		printf("%d:\t%.8f\t%.8f\n", i, h_result_gold[i], h_result[i]);
	}
	*/
	
	if (errCount > 0) {
		printf("\n@ERROR: TEST FAILED: %d results did not matched\n", errCount);
	}
	else if (zeroCount > 0){
		printf("\n@ERROR: TEST FAILED: %d results (from GPU) are zero\n", zeroCount);
	}
	else {
		printf("\nTEST PASSED: All results matched\n");
	}
	
	// Free-up device and host memory
	CUDA_SAFE_CALL(hipFree(d_x));
	CUDA_SAFE_CALL(hipFree(d_y));
	CUDA_SAFE_CALL(hipFree(d_result));
		   
	free(h_x);
	free(h_y);
	free(h_result);
		
	return 0;
}

void initializeArray1D(float *arr, int len, int seed) {
	int i;
	float randNum;
	srand(seed);

	for (i = 0; i < len; i++) {
		randNum = (float) rand();
		arr[i] = randNum;
	}
}
