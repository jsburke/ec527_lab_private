
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <math.h>
#include <time.h>

#define MINVAL 0.00
#define MAXVAL 10.0
#define TOL    1e-5
double CPS =   2.9e9;

int LEN;		// to be defined via cmd args

////////////////////////////  CUDA RELATED  ////////////////////////////////////


// Assertion to check for errors
#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"CUDA_SAFE_CALL: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__global__ void MMM_kernel(float* A, float* B, float* dst, int len)
{
	const int row = threadIdx.x + blockDim.x * blockIdx.x;
	const int col = threadIdx.y + blockDim.y * blockIdx.y;

	if(((row >= 0) && (row < len)) && ((col >= 0) && (col < len)))
	{
		int k;
		for(k = 0; k < len; k++) dst[row * len + col] = A[row * len + k] * B[k * len + col];
	}
}

//////////////////////////////  MATRIX  /////////////////////////////////////////

float* matrix_create(int len);
int    matrix_init(float* mat, int len);
int    matrix_zero(float* mat, int len);
int    matrix_copy(float* src, float* dst, int len);

/////////////////  Time related  //////////////////////////////

//rdtsc related
typedef union {
  unsigned long long int64;
  struct {unsigned int lo, hi;} int32;
} mcps_tctr;

#define MCPS_RDTSC(cpu_c) __asm__ __volatile__ ("rdtsc" : \
                     "=a" ((cpu_c).int32.lo), "=d"((cpu_c).int32.hi))

int clock_gettime(clockid_t clk_id, struct timespec *tp);
struct timespec diff(struct timespec start, struct timespec end);
double ts_ms(struct timespec ts);
struct timespec ts_diff(struct timespec start, struct timespec end);
double measure_cps(void);

////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char *argv[])
{

	return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////  MATRIX IMPLEMENTATIONS  ////////////////////////////////////////
float float_rand(float min, float max)
{
	float f = (float)random()/RAND_MAX;
	return  min + f * (max - min);
}


float* matrix_create(int len)
{
	float* arr;
	if(len > 0)
	{
		arr = (float*) calloc(len*len, sizeof(float));
		if(!arr)
		{
			printf("\n\tFailed to allocate array\n");
			return NULL;
		}
	}
	else return NULL;

	return arr;
}

int matrix_init(float* mat, int len)
{
	int len_sq, i;

	if(len > 0)
	{
		len_sq = len * len;
		for (i = 0; i < len_sq; i++)
		{
			mat[i] = float_rand(MINVAL, MAXVAL);
		}
		return 1;
	}
	printf("\nError in initializing matrix\n");
	return 0;
}

int matrix_zero(float* mat, int len)
{
	int len_sq, i;

	if(len > 0)
	{
		len_sq = len * len;
		for(i = 0; i < len_sq; i++)
		{
			mat[i] = 0;
		}
		return 1;
	}
	printf("\nFailed to zero matrix\n");
	return 0;
}

int   matrix_copy(float* src, float* dst, int len)
{
	int len_sq, i;

	if(len > 0)
	{
		len_sq = len * len;
		for(i = 0; i < len_sq; i++)
		{
			dst[i] = src[i];
		}
		return 1;
	}
	printf("\nFailed to copy matrix\n");
	return 0;
}

/////////////////////////////  Timing related  ///////////////////////////////

double ts_ms(struct timespec ts)
{
  return ((((double)(ts.tv_sec))*1.0e9) + ((double)(ts.tv_nsec)))/(1.0e6);
}

/* ---------------------------------------------------------------------------
| Make the CPU busy, and measure CPS (cycles per second).
|
| Explanation:
| If tests are very fast, they can run so quickly that the SpeedStep control
| (in kernel and/or on-chip) doesn't notice in time, and the first few tests
| might finish while the CPU is still in its sleep state (about 800 MHz,
| judging from my measurements)
|   A simple way to get around this is to run some kind of busy-loop that
| forces the OS and/or CPU to notice it needs to go to full clock speed.
| We print out the results of the computation so the loop won't get optimised
| away.
|
| Copy this code into other programs as desired. It provides three entry
| points:
|
| double ts_sec(ts): converts a timespec into seconds
| timespec ts_diff(ts1, ts2): computes interval between two timespecs
| measure_cps(): Does the busy loop and prints out measured CPS (cycles/sec)
--------------------------------------------------------------------------- */

struct timespec ts_diff(struct timespec start, struct timespec end)
{
  struct timespec temp;
  if ((end.tv_nsec-start.tv_nsec)<0) {
    temp.tv_sec = end.tv_sec-start.tv_sec-1;
    temp.tv_nsec = 1000000000+end.tv_nsec-start.tv_nsec;
  } else {
    temp.tv_sec = end.tv_sec-start.tv_sec;
    temp.tv_nsec = end.tv_nsec-start.tv_nsec;
  }
  return temp;
}

double measure_cps()
{
  struct timespec cal_start, cal_end;
  mcps_tctr tsc_start, tsc_end;
  double total_time;
  double total_cycles;
  /* We perform a chaotic iteration and print the result, to defeat
     compiler optimisation */
  double chaosC = -1.8464323952913974; double z = 0.0;
  long int i, ilim, j;

  /* Do it twice and throw away results from the first time; this ensures the
   * OS and CPU will notice it's busy and set the clock speed. */
  for(j=0; j<2; j++) {
    clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &cal_start);
    MCPS_RDTSC(tsc_start);
    ilim = 50*1000*1000;
    for (i=0; i<ilim; i++)
      z = z * z + chaosC;
    clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &cal_end);
    MCPS_RDTSC(tsc_end);
  }

  total_time = ts_ms(ts_diff(cal_start, cal_end));
  total_cycles = (double)(tsc_end.int64-tsc_start.int64);
  CPS = total_cycles / total_time;
  printf("z == %f, CPS == %g\n", z, CPS);

  return CPS;
}
/* ---------------------------------------------------------------------------
| End of measure_cps code
--------------------------------------------------------------------------- */

struct timespec diff(struct timespec start, struct timespec end)
{
  struct timespec temp;
  if ((end.tv_nsec-start.tv_nsec)<0) {
    temp.tv_sec = end.tv_sec-start.tv_sec-1;
    temp.tv_nsec = 1000000000+end.tv_nsec-start.tv_nsec;
  } else {
    temp.tv_sec = end.tv_sec-start.tv_sec;
    temp.tv_nsec = end.tv_nsec-start.tv_nsec;
  }
  return temp;
}
